// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>

#include "../../common/include/common.h"
#include "../../common/include/utils.cuh"
#include "../../common/include/concurrency.h"
#include "vectorized_pointwise.h"

namespace msamp {
void add_to_fp8(at::Tensor fp8_tensor,
                at::Tensor scale,
                at::Tensor scale_inv,
                at::Tensor amax,
                const at::Tensor& other,
                bool is_e4m3) {
  const size_t N = other.numel();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  TORCH_DTYPE_SWITCH_INPUT(other.scalar_type(), IType,
    SELECT_FP8_TYPE(is_e4m3, OType,
    
      constexpr int nvec = 32 / sizeof(IType);
      
      VectorizedAddToFp8KernelLauncher<nvec>(
        reinterpret_cast<IType*>(other.data_ptr()),
        reinterpret_cast<OType*>(fp8_tensor.data_ptr()),
        reinterpret_cast<fp32*>(scale.data_ptr()),
        reinterpret_cast<fp32*>(scale_inv.data_ptr()),
        reinterpret_cast<fp32*>(amax.data_ptr()),
        N,
        stream
      );
    );
  );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("add_to_fp8", &add_to_fp8, "Add to fp8");
}

} // namespace msamp
